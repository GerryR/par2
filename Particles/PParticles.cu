/**
* @file PParticles.cu
* @brief Implementation file for PParticles class.
*
* @author Calogero B. Rizzo
*
* @copyright This file is part of the PAR2 software.
*            Copyright (C) 2018 Calogero B. Rizzo
*
* @license This program is free software: you can redistribute it and/or modify
*          it under the terms of the GNU General Public License as published by
*          the Free Software Foundation, either version 3 of the License, or
*          (at your option) any later version.
*
*          This program is distributed in the hope that it will be useful,
*          but WITHOUT ANY WARRANTY; without even the implied warranty of
*          MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*          GNU General Public License for more details.
*
*          You should have received a copy of the GNU General Public License
*          along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "../Geometry/FaceField.cuh"
#include "../Geometry/CornerField.cuh"

#include <thrust/execution_policy.h>
#include <thrust/count.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <fstream>
#include <algorithm>

namespace par2
{
    template<typename T>
    struct InitCURAND
    {
        unsigned long long seed;
        hiprandState_t *states;
        InitCURAND(unsigned long long _seed, thrust::device_vector<hiprandState_t> &_states)
        {
            seed = _seed;
            states = thrust::raw_pointer_cast(_states.data());
        }

        __device__
        void operator()(unsigned int i)
        {
            hiprand_init(seed, i, 0, &states[i]);
        }
    };

    template<typename T>
    struct InitVolume
    {
        hiprandState_t* states;
        T p1x, p1y, p1z;
        T p2x, p2y, p2z;
        InitVolume(thrust::device_vector<hiprandState_t> &_states,
                    T _p1x, T _p1y, T _p1z,
                    T _p2x, T _p2y, T _p2z)
        {
            states = thrust::raw_pointer_cast(_states.data());
            p1x = _p1x;
            p1y = _p1y;
            p1z = _p1z;
            p2x = _p2x;
            p2y = _p2y;
            p2z = _p2z;
        }

        using Position = thrust::tuple<T, T, T>;

        __device__
        Position operator()(unsigned int i) const
        {
            Position p;

            thrust::get<0>(p) = p1x + (p2x-p1x)*hiprand_uniform(&states[i]);
            thrust::get<1>(p) = p1y + (p2y-p1y)*hiprand_uniform(&states[i]);
            thrust::get<2>(p) = p1z + (p2z-p1z)*hiprand_uniform(&states[i]);

            return p;
        }
    };

    template<typename T>
    PParticles<T>::PParticles(const grid::Grid<T> &_grid,
                           const thrust::host_vector<T> &_datax,
                           const thrust::host_vector<T> &_datay,
                           const thrust::host_vector<T> &_dataz,
                           T _molecularDiffusion,
                           T _alphaL,
                           T _alphaT,
                           unsigned int _nParticles,
                           long int _seed,
                           bool _useTrilinearCorrection)
            : nParticles(_nParticles), molecularDiffusion(_molecularDiffusion),
              alphaL(_alphaL), alphaT(_alphaT), grid(_grid), moveParticle(_grid),
              useTrilinearCorrection(_useTrilinearCorrection)
    {
        cx.resize(nParticles);
        cy.resize(nParticles);
        cz.resize(nParticles);

        datax = _datax;
        datay = _datay;
        dataz = _dataz;

        thrust::host_vector<T> _cdatax, _cdatay, _cdataz;
        if (useTrilinearCorrection)
        {
            par2::cornerfield::build(grid, _cdatax);
            par2::cornerfield::build(grid, _cdatay);
            par2::cornerfield::build(grid, _cdataz);

            par2::cornerfield::computeCornerVelocities(grid, _datax, _datay, _dataz,
                                                    _cdatax, _cdatay, _cdataz);
        }
        else
        {
            par2::cellfield::build(grid, _cdatax);
            par2::cellfield::build(grid, _cdatay);
            par2::cellfield::build(grid, _cdataz);

            par2::cellfield::computeDriftCorrection(grid, _datax, _datay, _dataz,
                                                    _cdatax, _cdatay, _cdataz,
                                                    molecularDiffusion, alphaL, alphaT);
        }
        cdatax = _cdatax;
        cdatay = _cdatay;
        cdataz = _cdataz;

        states.resize(maxParticles);
        thrust::counting_iterator<unsigned int> count(0);
        thrust::for_each(count, count+maxParticles, InitCURAND<T>(_seed, states));
        
        moveParticle.initialize(datax,
                                datay,
                                dataz,
                                cdatax,
                                cdatay,
                                cdataz,
                                molecularDiffusion,
                                alphaL,
                                alphaT,
                                states,
                                useTrilinearCorrection);

        hipDeviceSynchronize();

    }

    template<typename T>
    unsigned int PParticles<T>::size() const
    {
        return nParticles;
    }

    template<typename T>
    void PParticles<T>::initializeBox(T p1x, T p1y, T p1z,
                                      T p2x, T p2y, T p2z)
    {
        thrust::counting_iterator<unsigned int> count(0);
        auto pBeg = thrust::make_zip_iterator(
            thrust::make_tuple(cx.begin(), cy.begin(), cz.begin()));

        auto functor = InitVolume<T>(states, p1x, p1y, p1z, p2x, p2y, p2z);

        for (auto i = 0; i*maxParticles < nParticles; i++)
        {
            unsigned int kernelSize = maxParticles;
            if (kernelSize > nParticles - i*maxParticles)
            {
                kernelSize = nParticles - i*maxParticles;
            }
            thrust::transform(count,
                              count + kernelSize,
                              pBeg + i*maxParticles,
                              functor);
        }
    }

    template<typename T>
    void PParticles<T>::move(T dt)
    {
        thrust::counting_iterator<unsigned int> count(0);
        moveParticle.setTimeStep(dt);

        for (auto i = 0; i*maxParticles < nParticles; i++)
        {
            unsigned int kernelSize = maxParticles;
            if (kernelSize > nParticles - i*maxParticles)
            {
                kernelSize = nParticles - i*maxParticles;
            }

            auto pBeg = thrust::make_zip_iterator(
                thrust::make_tuple(cx.begin() + i*maxParticles,
                                   cy.begin() + i*maxParticles,
                                   cz.begin() + i*maxParticles,
                                   count));
            //auto pEnd = thrust::make_zip_iterator(
            //    thrust::make_tuple(cx.end(),   cy.end()  , cz.end()  , count+kernelSize));

            thrust::transform(pBeg, pBeg + kernelSize, pBeg, moveParticle);
        }
        hipDeviceSynchronize();

    }

    template<typename T>
    void PParticles<T>::exportCSV(const std::string &fileName) const
    {
        // Copy to host memory
        thrust::host_vector<T> hx = cx;
        thrust::host_vector<T> hy = cy;
        thrust::host_vector<T> hz = cz;

        std::ofstream outStream;
        outStream.open(fileName);
        if (outStream.is_open())
        {
            outStream << "id,x coord,y coord,z coord" << std::endl;
            for (unsigned int i = 0; i < nParticles; i++)
            {
                outStream << i << "," << hx[i] << "," << hy[i] << "," << hz[i]
                          << std::endl;
            }
        }
        else
        {
            throw std::runtime_error(std::string("Could not open file ") + fileName);
        }
        outStream.close();
    }

    template<typename T>
    struct isInside
    {
        T plane;

        T p1x, p1y, p1z;
        T p2x, p2y, p2z;
        isInside(T _p1x, T _p1y, T _p1z,
                 T _p2x, T _p2y, T _p2z)
        {
            p1x = _p1x;
            p1y = _p1y;
            p1z = _p1z;
            p2x = _p2x;
            p2y = _p2y;
            p2z = _p2z;
        }

        using Position = thrust::tuple<T, T, T>;

        __device__
        bool operator()(Position p) const
        {
            return (p1x <= thrust::get<0>(p) && thrust::get<0>(p) <= p2x) &&
                   (p1y <= thrust::get<1>(p) && thrust::get<1>(p) <= p2y) &&
                   (p1z <= thrust::get<2>(p) && thrust::get<2>(p) <= p2z);
        }
    };

    template<typename T>
    T PParticles<T>::concentrationBox(T p1x, T p1y, T p1z,
                                      T p2x, T p2y, T p2z) const
    {
        auto pBeg = thrust::make_zip_iterator(
            thrust::make_tuple(cx.begin(), cy.begin(), cz.begin()));
        auto pEnd = thrust::make_zip_iterator(
            thrust::make_tuple(cx.end(),   cy.end()  , cz.end()  ));

        return thrust::count_if(pBeg, pEnd,
                    isInside<T>(p1x, p1y, p1z, p2x, p2y, p2z))/T(nParticles);
    }

    template<typename T>
    struct isAfter
    {
        T plane;

        isAfter(T _plane) : plane(_plane) {};

        __device__
        bool operator()(T x)
        {
            return x > plane;
        }
    };

    template<typename T>
    T PParticles<T>::concentrationAfterX(T xplane) const
    {
        return thrust::count_if(cx.begin(), cx.end(),
                    isAfter<T>(xplane))/T(nParticles);
    }

}
