#include "hip/hip_runtime.h"
/**
* @file maingpu.cu
* @brief Entry point for PAR2
*
* @author Calogero B. Rizzo
*
* @copyright This file is part of the PAR2 software.
*            Copyright (C) 2018 Calogero B. Rizzo
*
* @license This program is free software: you can redistribute it and/or modify
*          it under the terms of the GNU General Public License as published by
*          the Free Software Foundation, either version 3 of the License, or
*          (at your option) any later version.
*
*          This program is distributed in the hope that it will be useful,
*          but WITHOUT ANY WARRANTY; without even the implied warranty of
*          MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*          GNU General Public License for more details.
*
*          You should have received a copy of the GNU General Public License
*          along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <iostream>
#include <iomanip>
#include <sstream>
#include <chrono>
#include <exception>
#include "Geometry/Point.cuh"
#include "Geometry/CartesianGrid.cuh"
#include "Geometry/FaceField.cuh"
#include "Geometry/CellField.cuh"
#include "Particles/PParticles.cuh"
#include "Utilities/Parameters.h"
#include "Utilities/spdlog/spdlog.h"

#include <thrust/host_vector.h>

namespace spd = spdlog;

void run(int argc, char** argv)
{
    auto console = spd::stdout_color_mt("console");
    console->set_pattern("%v");

    console->info("*********************************************************");
    console->info("*-------------------------------------------------------*");
    console->info("*------------------------- PAR2 ------------------------*");
    console->info("*-------------------------------------------------------*");
    console->info("*********************************************************");
    console->info("");

    std::string configurationFile;

    if (argc == 1)
    {
        console->info("Insert path to YAML configuration file:");
        std::cin >> configurationFile;
        console->info("");
    }
    else if (argc == 2)
    {
        configurationFile = std::string(argv[1]);
    }
    else
    {
        std::stringstream errorMessage;
        errorMessage << "Specify path to the YAML configuration file: "
                     << "(> par2 /path/to/configuration.yaml)";
        throw std::runtime_error(errorMessage.str());
    }

    std::string configurationPath =
        configurationFile.substr(0, configurationFile.find_last_of("/\\") + 1);

    auto logfile = spd::basic_logger_mt("logfile",
                configurationPath + std::string("par2.log"), true);

    logfile->info("Logfile created {}", configurationFile);

    // Start time
    auto t_start = std::chrono::system_clock::now();

    // Load configuration file
    par2::Parameters<PAR2_FLOAT> par(configurationFile);

    // Create grid
    auto grid = par2::grid::build<PAR2_FLOAT>(par.nx(), par.ny(), par.nz(),
                                         par.dx(), par.dy(), par.dz());
    logfile->info("Grid size: {} x {} x {}", par.nx(), par.ny(), par.nz());
    logfile->info("Cell size: {} x {} x {}", par.dx(), par.dy(), par.dz());

    // Create and load velocity field
    console->info("Import from MODFLOW...");
    thrust::host_vector<PAR2_FLOAT> datax, datay, dataz;
    par2::facefield::build<PAR2_FLOAT>(grid, datax, datay, dataz);

    if (par.velType() == "modflow")
    {
        par2::facefield::importFromModflow(grid, datax, datay, dataz,
                                          configurationPath + par.velPath(),
                                          par.rho());
        logfile->info("FTL file: {}", configurationPath + par.velPath());
        logfile->info("Porosity: {}", par.rho());
    }
    else
    {
        throw par2::yaml_invalid_argument("Velocity type must be 'modflow'.");
    }

    // Choose interpolation method
    bool cTrilinear = false;
    if (par.interp() == "trilinear")
    {
        cTrilinear = true;
    }
    else if (par.interp() == "finite difference")
    {
        cTrilinear = false;
    }
    else
    {
        throw par2::yaml_invalid_argument("Interpolation must be either 'trilinear' or 'finite difference'.");
    }
    logfile->info("Trilinear: {}", cTrilinear);

    // Create and initialize particles
    console->info("Device initialization...");
    long int seed = par.seed();
    logfile->info("Seed: {}", seed);
    logfile->info("Molecular diffusion: {}", par.Dm());
    logfile->info("Longitudinal dispersivity: {}", par.alphaL());
    logfile->info("Transverse dispersivity: {}", par.alphaT());
    logfile->info("Particles: {}", par.nParticles());
    par2::PParticles<PAR2_FLOAT> particles(grid, datax, datay, dataz,
                                     par.Dm(), par.alphaL(), par.alphaT(),
                                     par.nParticles(), seed, cTrilinear);

    logfile->info("Box P1: ({}, {}, {})", par.p1x(), par.p1y(), par.p1z());
    logfile->info("Box P2: ({}, {}, {})", par.p2x(), par.p2y(), par.p2z());
    particles.initializeBox(par.p1x(), par.p1y(), par.p1z(),
                            par.p2x(), par.p2y(), par.p2z());

    // Initialize CSV output files
    std::ofstream csvOutStream;
    if (par.csvOutput())
    {
        csvOutStream.open(configurationPath + par.csvPath());

        // Precision of the output
        csvOutStream << std::setprecision(15) << std::fixed;

        if (csvOutStream.is_open())
        {
            console->info("Prepare output ({})...", configurationPath + par.csvPath());

            // Write labels in the first row
            // First two columns contain the current step and time
            csvOutStream << "step, time";
            for (auto i = 0; i < par.csvNumberOfItems(); i++)
            {
                csvOutStream << ", " << par.csvItemLabel(i);
            }
            csvOutStream << std::endl;

            logfile->info("{} ready", par.csvPath());
        }
        else
        {
            throw std::runtime_error(std::string("Could not open file ") +
                        configurationPath + par.csvPath());
        }
    }

    // Keep track of the next step where we want to output a full particles
    // snapshot. If stepSnapshot=-1, no snapshots will be created.
    int stepSnapshot = -1;
    int stepId       = -1;

    if (par.snapshotOutput())
    {
        if (par.snapshotUseSkip())
        {
            stepSnapshot = 0;
        }
        else if (par.snapshotSize() > 0)
        {
            stepId = 0;
            stepSnapshot = par.snapshotStep(stepId);
        }
    }

    // Ready time
    auto t_ready = std::chrono::system_clock::now();

    // Simulation variables
    int steps = par.steps();
    PAR2_FLOAT dt = par.dt();
    PAR2_FLOAT completed = 0.0;

    console->info("");
    console->info("Start simulation...");
    // Start time loop
    // Using steps+1 to take into account the last step.
    for (auto step = 0; step < steps+1; step++)
    {
        // CSV Output
        if (par.csvOutput())
        {
            if (step%par.csvSkip() == 0)
            {
                logfile->info("Writing CSV file (STEP {})", step);

                // Write step and time
                csvOutStream << step << ", " << step*dt;
                for (auto i = 0; i < par.csvNumberOfItems(); i++)
                {
                    csvOutStream << ", ";
                    if (!par.csvItemType(i).compare("after-x"))
                    {
                        csvOutStream << particles.concentrationAfterX(par.csvItemX(i));
                    }
                    else if (!par.csvItemType(i).compare("box"))
                    {
                        csvOutStream << particles.concentrationBox(
                                            par.csvItemP1X(i),
                                            par.csvItemP1Y(i),
                                            par.csvItemP1Z(i),
                                            par.csvItemP2X(i),
                                            par.csvItemP2Y(i),
                                            par.csvItemP2Z(i));
                    }
                    else
                    {
                        throw par2::yaml_invalid_argument("CSV Item type must be either 'after-x' or 'box'.");
                    }
                }
                csvOutStream << std::endl;
            }
        }

        // Snapshot Output
        if (step == stepSnapshot)
        {
            // Export particle positions
            auto snapshotOutputPath = configurationPath + par.snapshotPath(step);

            logfile->info("Writing Snapshot file {} (STEP {})", snapshotOutputPath, step);
            console->info("Writing Snapshot file {}", snapshotOutputPath);

            particles.exportCSV(snapshotOutputPath);

            // Update next snapshot
            if (par.snapshotUseSkip())
            {
                stepSnapshot += par.snapshotSkip();
            }
            else
            {
                stepId++;
                if (stepId == par.snapshotSize())
                {
                    stepSnapshot = -1;
                }
                else
                {
                    stepSnapshot = par.snapshotStep(stepId);
                }
            }
        }

        // Move particles
        particles.move(dt);

        if (step >= completed)
        {
            console->info("{:3.0f}% completed (STEP {})", completed/steps*100.0, step);
            completed += steps/50.0;
        }
    }

    console->info("Simulation completed");
    console->info("");

    // Close CSV file
    if (par.csvOutput())
    {
        csvOutStream.close();
    }

    // End time
    auto t_end = std::chrono::system_clock::now();

    // Print information about computation time
    std::chrono::duration<PAR2_FLOAT> preproc_seconds    = t_ready - t_start;
    std::chrono::duration<PAR2_FLOAT> simulation_seconds = t_end - t_ready;
    std::chrono::duration<PAR2_FLOAT> elapsed_seconds    = t_end - t_start;

    logfile->info("TIME PREPROCESSING: {}s", preproc_seconds.count());
    logfile->info("TIME SIMULATION: {}s", simulation_seconds.count());
    logfile->info("TIME ELAPSED: {}s", elapsed_seconds.count());

    console->info("TIME PREPROCESSING: {}s", preproc_seconds.count());
    console->info("TIME SIMULATION: {}s", simulation_seconds.count());
    console->info("TIME ELAPSED: {}s", elapsed_seconds.count());
}

int main(int argc, char** argv)
{
    int EXIT = EXIT_SUCCESS;

    try
    {
        run(argc, argv);
    }
    catch (const std::exception& e)
    {
        auto console = spdlog::get("console");
        if (console)
        {
            console->critical("Execution terminated with an error:");
            console->critical(e.what());
        }
        auto logfile = spdlog::get("logfile");
        if (logfile)
        {
            logfile->critical("Execution terminated with an error:");
            logfile->critical(e.what());
        }
        EXIT = EXIT_FAILURE;
    }
    // Release and close all loggers
    spd::drop_all();

    return EXIT;
}
